#include "hip/hip_runtime.h"
/*================================================================
 * Laplacian filter
 *----------------------------------------------------------------
 * No licence, public domain.
 *
 * 
 *
 * Um6ra1
 *
 *================================================================*/

#include "Laplacian.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define LAPNUM	9
__constant__ int gc_weight[LAPNUM];

__global__ static void KerLaplacian(UINT32 *lpDst, UINT32 *lpSrc, int width, int height, int amplitude);

__device__ int KerSobel(int a1, int a2, int a3, int a4, int a5, int a6)
{
	return(a1 + 2 * a2 + a3 - (a4 + 2 * a5 + a6));
}

__global__ static void KerLaplacian(UINT32 *lpDst, UINT32 *lpSrc, int width, int height, int amplitude)
{
	int	x	= blockIdx.x * blockDim.x + threadIdx.x;
	int	y	= blockIdx.y * blockDim.y + threadIdx.y;
	int idx	= width * y + x;
	int	xy[9];
	int dr, dg, db;
	int powR, powG, powB;

	if(x >= 1 && x < width - 1 && y >= 1 && y < height - 1)
	{
		/*lpDst[idx] = 8 * lpSrc[idx] -
			lpSrc[idx - 1] - lpSrc[idx + 1] -
			lpSrc[idx - width] - lpSrc[idx + width] -
			lpSrc[idx - width - 1] - lpSrc[idx + width + 1] -
			lpSrc[idx - width + 1] - lpSrc[idx + width - 1];
		*/
		
		xy[0]	= lpSrc[idx - width - 1];
		xy[1]	= lpSrc[idx - width];
		xy[2]	= lpSrc[idx - width + 1];
		xy[3]	= lpSrc[idx - 1];
		xy[4]	= lpSrc[idx];
		xy[5]	= lpSrc[idx + 1];
		xy[6]	= lpSrc[idx + width - 1];
		xy[7]	= lpSrc[idx + width];
		xy[8]	= lpSrc[idx + width + 1];

		dr = dg = db = 0;

		for(int i = 0; i < 9; i ++)
		{
			dr += gc_weight[i] * (0xFF & (xy[i] >> 16));
			dg += gc_weight[i] * (0xFF & (xy[i] >> 8));
			db += gc_weight[i] * (0xFF & (xy[i]));
		}

		/* Calculate power */
		powR = amplitude * dr * dr >> 10;	// * amplitude / 1024
		powG = amplitude * dg * dg >> 10;
		powB = amplitude * db * db >> 10;
		if(powR > 255)	{	powR = 255;	}
		if(powG > 255)	{	powG = 255;	}
		if(powB > 255)	{	powB = 255;	}

		lpDst[y * width + x]	= (powR << 16) | (powG << 8) | (powB);
	}
}

#define BLOCKSIZE	16
#include <cstdio>

void Imgproc::CuLaplacian(UINT32 *lpDst, UINT32 *lpSrc, int width, int height, int amplitude)
{
	static int	weight[] = // Laplacian kernel
	{
		-1, -1, -1,
		-1,  8, -1,
		-1, -1, -1
	};
	UINT32	*d_lpSrc, *d_lpDst;
//	int	nThreadsPerBlock;
//	int	nBlocksPerGrid;

	::hipMemcpyToSymbol(HIP_SYMBOL(gc_weight), weight, sizeof(weight));
	::hipMalloc((void **)&d_lpSrc, sizeof(UINT32) * width * height);
	::hipMalloc((void **)&d_lpDst, sizeof(UINT32) * width * height);
	::hipMemcpy(d_lpSrc, lpSrc, sizeof(UINT32) * width * height, hipMemcpyHostToDevice);

	dim3	dimThread(BLOCKSIZE, BLOCKSIZE);
	dim3	dimBlock((width + BLOCKSIZE - 1) / BLOCKSIZE, (height + BLOCKSIZE - 1) / BLOCKSIZE);

	KerLaplacian<<<dimBlock, dimThread>>>(d_lpDst, d_lpSrc, width, height, amplitude);

	::hipMemcpy(lpDst, d_lpDst, sizeof(UINT32) * width * height, hipMemcpyDeviceToHost);

	::hipFree(d_lpSrc);
	::hipFree(d_lpDst);
}